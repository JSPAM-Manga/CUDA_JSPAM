#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



#include <string>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <algorithm> 
#include <string>
#include <vector>
#include <sstream>
#include <functional>
#include <time.h> 
#include <cmath>
#include <climits>
#include <memory.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

# define M_PI   3.14159265358979323846
# define G1n	1000
# define G2n	1000
# define Gn	G1n+G2n+1
# define DF_nnn 2000
# define theta_min 0
# define theta_max 180
# define theta_step 5
# define phi_min 0
# define phi_max 360
# define phi_step 5

using namespace std;

struct tmins
{
	double t, min_dist, min_vel, rv7;
};
struct coe
{
	double e, a, i, o, w, v;
};
struct vec
{
	double x, y, z;
	__device__ vec()
	{
		x = y = z = 0;
	}
	__device__ vec(double ix, double iy, double iz)
	{
		x = ix;
		y = iy;
		z = iz;
	}
	__device__ inline vec operator+(vec a) {
		vec r;
		r.x = a.x + x;
		r.y = a.y + y;
		r.z = a.z + z;
		return r;
	}
	__device__ inline vec operator-() {
		vec r;
		r.x = -x;
		r.y = -y;
		r.z = -z;
		return r;
	}
	__device__ inline vec operator-(vec a) {
		vec r;
		r = *this + -a;
		return r;
	}
	__device__ double dist() {
		return sqrt(pow(x, 2) + pow(y, 2) + pow(z, 2));
	}
	__device__ double sqrd() {
		return (pow(x, 2) + pow(y, 2) + pow(z, 2));
	}
	__device__ double dot(vec a) {
		return (a.x*x + a.y*y + a.z*z);
	}
	__device__ double mag(vec a) {
		return sqrt(this->dot(a));
	}
	__device__ vec cross(vec a) {
		vec r;
		r.x = y*a.z - z*a.y;
		r.y = z*a.x - x*a.z;
		r.z = x*a.y - y*a.x;
		return r;
	}
	__device__ vec scale(double sc) {
		vec r;
		r = *this*sc;
		return r;
	}
	__device__ inline vec operator*(vec a) {
		vec r;
		r.x = a.x * x;
		r.y = a.y * y;
		r.z = a.z * z;
		return r;
	}
	__device__ inline vec operator*(double a) {
		vec r;
		r.x = a * x;
		r.y = a * y;
		r.z = a * z;
		return r;
	}
	__device__ inline vec operator/(double a) {
		vec r;
		r.x = x / a;
		r.y = y / a;
		r.z = z / a;
		return r;
	}
	__device__ inline vec operator=(double a) {
		vec r;
		r.x = a;
		r.y = a;
		r.z = a;
		return r;
	}
};
struct pos_vel
{
	vec r;
	vec v;
	__device__ inline pos_vel operator+(pos_vel a) {
		pos_vel ret;
		ret.r = a.r + r;
		ret.v = a.v + v;
		return ret;
	}
	__device__ inline pos_vel operator*(pos_vel a) {
		pos_vel ret;
		ret.r = a.v * r;
		ret.v = a.v * v;
		return ret;
	}
	__device__ inline pos_vel operator*(double a) {
		pos_vel ret;
		ret.r = r * a;
		ret.v = v * a;
		return ret;
	}
	__device__ inline pos_vel operator/(double a) {
		pos_vel ret;
		ret.r = r / a;
		ret.v = v / a;
		return ret;
	}
};
struct gparam
{
	int galaxy;
	double mass, eps, epsilon, rin, rout, theta, phi, heat;
	vec rscale;
	int opt;
	int n;
};
class parameters
{
public:
	double mass_gm = 1.98892e44;
	double mass_solar = 1.98892e33;
	double distance = 4.6285203749999994e22;
	double time_s = 2.733342473337471e15;
	double vel_unit = distance / time_s;
	double pc = 3.08568025e18;
	double kpc = pc * 1000.0;
	double year = 365.25 * 24.0 * 3600.0;
	double km = 1e5;
	double vel_km_sec = vel_unit / km;
	double a_mss = distance / (time_s * time_s) / 100.0;
	double a0_mks = 1.2e-10;
	double a0 = a0_mks / a_mss;
	double pi = 3.141592653589793;
	double hbase = 0.001;

	int potential_type = 0;

	int ndim = 3;

	gparam galaxy1, galaxy2;

	pos_vel x0[Gn], xout[Gn];
	//double** x0, xout;

	int n;

	double time, tstart, tend;
	double inclination_degree;
	double omega_degree;
	double rmin;
	double velocity_factor;
	double mratio;
	double secondary_size;
	pos_vel sec_vec;
	bool use_sec_vec, tIsSet;

	double h;
	int nstep;
	int nout;

	int iout;
	int unit;
	int istep;

	hiprandState state;

	__device__ void standard_galaxy_both()
	{
		standard_galaxy(galaxy1, 1);
		standard_galaxy(galaxy2, 2);
	}

	__device__ void standard_galaxy(gparam& g, int galaxy)
	{
		g.galaxy = galaxy;
		g.mass = 1.0;
		g.epsilon = 0.3;
		g.rin = 0.05;
		g.rout = 1.0;
		g.rscale.x = 3.0;
		g.rscale.y = 3.0;
		g.rscale.z = 3.0;
		g.theta = 0.0;
		g.phi = 0.0;
		g.opt = 1;
		g.heat = 0.0;
		g.n = 1000;
	}

	__device__ void test_collision() {
		inclination_degree = 90.0;
		omega_degree = 0.0;
		rmin = 1.0;
		velocity_factor = 1.0;
		time = -3.0;

		h = hbase;
		nout = 5;
		nstep = 500;

		n = galaxy1.n + galaxy2.n;
	}

	__device__ vec unrotate_frame(vec in, double stheta, double ctheta, double sphi, double cphi) {
		vec r;

		r.x = in.x * ctheta + in.z * stheta;
		r.y = in.y;
		r.z = -in.x * stheta + in.z * ctheta;

		r.x = r.x * cphi - r.y * sphi;
		r.y = r.x * sphi + r.y * cphi;

		return r;
	}

	__device__ vec rotate_frame(vec in, double stheta, double ctheta, double sphi, double cphi) {
		vec r;

		r.x = in.x * cphi + in.y * sphi;
		r.y = -in.x * sphi + in.y * cphi;
		r.z = in.z;

		r.x = r.x * ctheta - r.z * stheta;
		r.z = r.x * stheta + r.z * ctheta;

		return r;
	}
	void input_particles(ifstream stream) {
		for (int i = 0; i < n; i++) {
			stream >> x0[i].r.x;
			stream >> x0[i].r.y;
			stream >> x0[i].r.z;
			stream >> x0[i].v.x;
			stream >> x0[i].v.y;
			stream >> x0[i].v.z;

		}

	}
	void output_particles(FILE* file, bool header_on) {
		if (header_on)
		{
			//fprintf(file, "%16.8f\n", time);
			//fprintf(file, "%16.8f%16.8f\n", galaxy1.mass, galaxy2.mass);
			//fprintf(file, "%16.8f%16.8f\n", galaxy1.eps, galaxy2.eps);
			fprintf(file, "n:%8i n1:%8i n2:%8i theta1:%16.8f phi1:%16.8f\n", n, galaxy1.n, galaxy2.n, galaxy1.theta, galaxy1.phi);
		}
		for (int i = 0; i < n; i++)
			fprintf(file, "%16.8f%16.8f%16.8f%16.8f%16.8f%16.8f\n", x0[i].r.x, x0[i].r.y, x0[i].r.z, x0[i].v.x, x0[i].v.y, x0[i].v.z);
	}
	void create_gnuplot_script()
	{
		FILE *file;
		double xmin, xmax, ymin, ymax;
		double amax;
		//xmin = *min_element(x0[1], x0[1] + iout);
		//xmax = *max_element(x0[1], x0[1] + iout);
		//ymin = *min_element(x0[2], x0[2] + iout);
		//ymax = *max_element(x0[2], x0[2] + iout);
		xmin = minx(x0, iout);
		xmax = maxx(x0, iout);
		ymin = miny(x0, iout);
		ymax = maxy(x0, iout);

		amax = max(-xmin, xmax);
		amax = max(amax, -ymin);
		amax = max(amax, ymax);

		file = fopen("gscript", "w+");
		fprintf(file, "%s%15.6f%s%15.6f%s", "set xrange[,", -amax, ":", amax, "]");
		fprintf(file, "%s%15.6f%s%15.6f%s", "set yrange[,", -amax, ":", amax, "]");
		for (int i = 0; i < iout; i++)
			fprintf(file, "%s%3i%s", "plot 'a.", i, "' using 1:2");
		fclose(file);
	}
	double minx(pos_vel* pv, int n)
	{
		double min = INT_MAX;
		for (int i = 0; i < n; i++)
			if (pv[i].r.x < min)
				min = pv[i].r.x;
		return min;
	}
	double miny(pos_vel* pv, int n)
	{
		double min = INT_MAX;
		for (int i = 0; i < n; i++)
			if (pv[i].r.y < min)
				min = pv[i].r.y;
		return min;
	}
	double maxx(pos_vel* pv, int n)
	{
		double max = INT_MIN;
		for (int i = 0; i < n; i++)
			if (pv[i].r.x > max)
				max = pv[i].r.x;
		return max;
	}
	double maxy(pos_vel* pv, int n)
	{
		double max = INT_MIN;
		for (int i = 0; i < n; i++)
			if (pv[i].r.y > max)
				max = pv[i].r.y;
		return max;
	}
	void print_profile(int galaxy_num)
	{
		gparam g;
		if (galaxy_num == 1)
			g = galaxy1;
		else
			g = galaxy2;

		cout << "----------------------------------";
		cout << "GALAXY =" << g.galaxy;
		cout << "mass        = " << g.mass;
		cout << "epsilon     = " << g.eps;
		cout << "rin         = " << g.rin;
		cout << "rout        = " << g.rout;
		cout << "rscale      = " << g.rscale.x;
		cout << "rscale      = " << g.rscale.y;
		cout << "rscale      = " << g.rscale.z;
		cout << "theta       = " << g.theta;
		cout << "phi         = " << g.phi;
		cout << "opt         = " << g.opt;
		cout << "heat        = " << g.heat;
		cout << "particles   = " << g.n;
		cout << "----------------------------------";
	}
	void print_collision()
	{
		cout << "----------------------------------";
		cout << "COLLISION PARAMETERS";
		cout << "n           = " << n;
		cout << "time        = " << time;
		cout << "inclination = " << inclination_degree;
		cout << "omega       = " << omega_degree;
		cout << "rmin        = " << rmin;
		cout << "velocity    = " << velocity_factor;
		cout << "h           = " << h;
		cout << "nstep       = " << nstep;
		cout << "nout        = " << nout;
		cout << "----------------------------------";
	}
	void octave_parameters_out(pos_vel pv, pos_vel x00)
	{
		cout << "$mass1 = " << galaxy1.mass << ";" << endl;
		cout << "$t1    = " << galaxy1.theta << ";" << endl;
		cout << "$p1    = " << galaxy1.phi << ";" << endl;
		cout << "$rout1 = " << galaxy1.rout << ";" << endl;
		cout << "$mass2 = " << galaxy2.mass << ";" << endl;
		cout << "$t2    = " << galaxy2.theta << ";" << endl;
		cout << "$p2    = " << galaxy2.phi << ";" << endl;
		cout << "$rout2 = " << galaxy2.rout << ";" << endl;
		cout << "$xf    = " << pv.r.x << ";" << endl;
		cout << "$yf    = " << pv.r.y << ";" << endl;
		cout << "$zf    = " << pv.r.z << ";" << endl;
		cout << "$vxf   = " << pv.v.x << ";" << endl;
		cout << "$vyf   = " << pv.v.y << ";" << endl;
		cout << "$vzf   = " << pv.v.z << ";" << endl;
		cout << "$x     = " << x00.r.x << ";" << endl;
		cout << "$y     = " << x00.r.y << ";" << endl;
		cout << "$z     = " << x00.r.z << ";" << endl;
		cout << "$vx    = " << x00.v.x << ";" << endl;
		cout << "$vy    = " << x00.v.y << ";" << endl;
		cout << "$vz    = " << x00.v.z << ";" << endl;
		cout << "$t     = " << tend << ";" << endl;
	}
	void read_parameter_file(ifstream& in)
	{
		string line, label;
		double val;
		while (!in.eof()) {
			line.clear();
			label.clear();
			val = 0;
			in >> line;
			split_str(line, label, val);
			if (label == "potential_type")
				potential_type = (int)val;
			else if (label == "mass1")
				galaxy1.mass = val;
			else if (label == "mass2")
				galaxy2.mass = val;
			else if (label == "epsilon1")
				galaxy1.epsilon = val;
			else if (label == "epsilon2")
				galaxy2.epsilon = val;
			else if (label == "rin1")
				galaxy1.rin = val;
			else if (label == "rin2")
				galaxy2.rin = val;
			else if (label == "rout1")
				galaxy1.rout = val;
			else if (label == "rout2")
				galaxy2.rout = val;
			else if (label == "theta1")
				galaxy1.theta = val;
			else if (label == "theta2")
				galaxy2.theta = val;
			else if (label == "phi1")
				galaxy1.phi = val;
			else if (label == "phi2")
				galaxy2.phi = val;
			else if (label == "opt1")
				galaxy1.opt = (int)val;
			else if (label == "opt2")
				galaxy2.opt = (int)val;
			else if (label == "heat1")
				galaxy1.heat = val;
			else if (label == "heat2")
				galaxy2.heat = val;
			else if (label == "n1")
				galaxy1.n = (int)val;
			else if (label == "n2")
				galaxy2.n = (int)val;
			else if (label == "inclination_degree")
				inclination_degree = val;
			else if (label == "omega_degree")
				omega_degree = val;
			else if (label == "rmin")
				rmin = val;
			else if (label == "velocity_factor")
				velocity_factor = val;
			else if (label == "tstart") {
				time = val;
				tstart = val;
				tIsSet = true;
			}
			else if (label == "tend")
				tend = val;
			else if (label == "h")
				h = val;
			else if (label == "rx") {
				use_sec_vec = true;
				sec_vec.r.x = val;
			}
			else if (label == "ry") {
				use_sec_vec = true;
				sec_vec.r.y = val;
			}
			else if (label == "rz") {
				use_sec_vec = true;
				sec_vec.r.z = val;
			}
			else if (label == "vx") {
				use_sec_vec = true;
				sec_vec.v.x = val;
			}
			else if (label == "vy") {
				use_sec_vec = true;
				sec_vec.v.y = val;
			}
			else if (label == "vz") {
				use_sec_vec = true;
				sec_vec.v.z = val;
			}
			else if (label == "rscale11")
				galaxy1.rscale.x = val;
			else if (label == "rscale12")
				galaxy1.rscale.y = val;
			else if (label == "rscale13")
				galaxy1.rscale.z = val;
			else if (label == "rscale21")
				galaxy2.rscale.x = val;
			else if (label == "rscale22")
				galaxy2.rscale.y = val;
			else if (label == "rscale23")
				galaxy2.rscale.z = val;
			else
				cout << "skipping line ";
		}
	}
	void split_str(string in, string& label, double& val)
	{
		string sval;
		int ind, len;
		char strt = in.at(0);
		if (strt == '!' || strt == '#' || strt == '/') {
			label = "!";
			val = 0;
			return;
		}
		ind = in.find("=");
		if (ind == 0) {
			label = "!";
			val = 0;
		}
		label = in.substr(0, ind);
		len = in.length() - label.length();
		sval = in.substr(ind + 1, len);
		val = strtod(sval.c_str(), NULL);
	}

	void write_parameter_file(ofstream o)
	{
		o << "potential_type=" << potential_type;
		o << "mass1=" << galaxy1.mass;
		o << "mass2=" << galaxy2.mass;
		o << "epsilon1=" << galaxy1.epsilon;
		o << "epsilon2=" << galaxy2.epsilon;
		o << "rin1=" << galaxy1.rin;
		o << "rin2=" << galaxy2.rin;
		o << "rout1=" << galaxy1.rout;
		o << "rout2=" << galaxy2.rout;
		o << "theta1=" << galaxy1.theta;
		o << "theta2=" << galaxy2.theta;
		o << "phi1=" << galaxy1.phi;
		o << "phi2=" << galaxy2.phi;
		o << "opt1=" << galaxy1.opt;
		o << "opt2=" << galaxy2.opt;
		o << "heat1=" << galaxy1.heat;
		o << "heat2=" << galaxy2.heat;
		o << "n1=" << galaxy1.n;
		o << "n2=" << galaxy2.n;
		o << "inclination_degree=" << inclination_degree;
		o << "omega_degree=" << omega_degree;
		o << "rmin=" << rmin;
		o << "velocity_factor=" << velocity_factor;
		o << "tstart=" << tstart;
		o << "tend=" << tend;
		o << "h=" << h;
		o << "rx=" << sec_vec.r.x;
		o << "ry=" << sec_vec.r.y;
		o << "rz=" << sec_vec.r.z;
		o << "vx=" << sec_vec.v.x;
		o << "vy=" << sec_vec.v.y;
		o << "vz=" << sec_vec.v.z;
		o << "rscale11=" << galaxy1.rscale.x;
		o << "rscale12=" << galaxy1.rscale.y;
		o << "rscale13=" << galaxy1.rscale.z;
		o << "rscale21=" << galaxy2.rscale.x;
		o << "rscale22=" << galaxy2.rscale.y;
		o << "rscale23=" << galaxy2.rscale.z;
	}

	__device__ void set_state_info(double* infos)
	{
		potential_type = 0;
		sec_vec.r.x = infos[1];
		sec_vec.r.y = infos[2];
		sec_vec.r.z = infos[3];
		sec_vec.v.x = infos[4];
		sec_vec.v.y = infos[5];
		sec_vec.v.z = infos[6];
		galaxy1.mass = infos[7];
		galaxy2.mass = infos[8];
		galaxy1.rout = infos[9];
		galaxy2.rout = infos[10];
		galaxy1.phi = infos[11];
		galaxy2.phi = infos[12];
		galaxy1.theta = infos[13];
		galaxy2.theta = infos[14];
		galaxy1.epsilon = infos[15];
		galaxy2.epsilon = infos[16];
		galaxy1.rscale.x = infos[17];
		galaxy1.rscale.y = infos[18];
		galaxy1.rscale.z = infos[19];
		galaxy2.rscale.x = infos[20];
		galaxy2.rscale.y = infos[21];
		galaxy2.rscale.z = infos[22];
		use_sec_vec = true;
	}

	__device__ void parse_state_info_string(string in)
	{
		istringstream ss(in);
		string token;
		double infos[23];
		int i = 0;
		while (getline(ss, token, ',')) {
			infos[i] = strtod(token.c_str(), NULL);
		}
		potential_type = 0;
		sec_vec.r.x = infos[1];
		sec_vec.r.y = infos[2];
		sec_vec.r.z = infos[3];
		sec_vec.v.x = infos[4];
		sec_vec.v.y = infos[5];
		sec_vec.v.z = infos[6];
		galaxy1.mass = infos[7];
		galaxy2.mass = infos[8];
		galaxy1.rout = infos[9];
		galaxy2.rout = infos[10];
		galaxy1.phi = infos[11];
		galaxy2.phi = infos[12];
		galaxy1.theta = infos[13];
		galaxy2.theta = infos[14];
		galaxy1.epsilon = infos[15];
		galaxy2.epsilon = infos[16];
		galaxy1.rscale.x = infos[17];
		galaxy1.rscale.y = infos[18];
		galaxy1.rscale.z = infos[19];
		galaxy2.rscale.x = infos[20];
		galaxy2.rscale.y = infos[21];
		galaxy2.rscale.z = infos[22];
		use_sec_vec = true;
	}
};
class df_module
{
public:
	const static int nnn = DF_nnn;
	double rad[nnn];
	double rho_halo[nnn], mass_halo[nnn];
	double rho_disk[nnn], mass_disk[nnn];
	double rho_bulge[nnn], mass_bulge[nnn];
	double rho_total[nnn], mass_total[nnn];
	double masses[nnn], radius[nnn], density[nnn];
	double vr2[nnn], vr[nnn], new_vr2[nnn], new_vr[nnn];
	double acceleration[nnn], acceleration_particle[nnn];
	double new_mass[nnn], new_rho[nnn], phi[nnn];

	double rs_internal = 10.0;

	double rs2 = rs_internal * rs_internal;
	double rs3 = rs2 * rs_internal;

	double pscale;
	double lnl;

	__device__ void init_distribution()
	{
		double rmax;
		double mold, dmold, mtot;
		double rscale;
		double dx, x;
		double alphahalo, qhalo, gammahalo, mhalo, rchalo, rhalo, epsilon_halo;
		double zdisk, hdisk, zdiskmax;
		double hbulge, mbulge;
		double rho_tmp;
		double G, factor;
		double r, m, sqrtpi;
		double p1, rd, rho_local;
		double p, rr, dr, rh, dp, mnew, dm;
		double acc_merge, rad_merge, acc;
		double pi = M_PI;

		int j, nmax, k, nmerge, ntotal, jj;

		//set the constant for dynamical friction
		//lnl = 0.00;
		//default for Merger Zoo
		lnl = 0.001;

		//set up the parameters for the halo
		mhalo = 5.8;
		rhalo = 10.0;
		rchalo = 10.0;
		gammahalo = 1.0;
		epsilon_halo = 0.4;
		sqrtpi = sqrt(pi);
		//////////
		//derive additional constants
		qhalo = gammahalo / rchalo;
		alphahalo = 1.0 / (1.0 - sqrtpi * qhalo * exp(pow(qhalo, 2)) * (1.0 - erf(qhalo)));
		//////////
		//set the integration limits and zero integration constants
		rmax = 20;
		nmax = 2000;
		dr = rmax / (nmax);
		mold = 0;
		rscale = 5;
		//ntotal = nmax * rscale;
		ntotal = nnn;
		//////////
		//set the limits for integration, and zero integration constants
		k = nmax / 2;
		dx = 1.0 / k;
		x = 0.0;
		dmold = 0.0;
		mtot = 0.0;
		//rad = 0.0;
		memset(rad, 0, 4 * nnn);
		m = 0.0;
		G = 1;
		//////////
		//set the fundamental disk parameters
		zdisk = 0.2;
		zdiskmax = 3.5;
		hdisk = 1.0;
		//////////
		//set the fundamental bulge parameters
		hbulge = 0.2;
		mbulge = 0.3333;
		//////////////////////////////////////////////////////////////////////////////////////////////
		//////////set up the radius array
		for (j = 0; j < nmax; j++) {
			x = x + dx;
			rad[j] = x * rchalo;
		}
		//////////////////////////////////////////////////////////////////////////////////////////////
		//////////
		dr = rad[2] - rad[1];
		dx = dr / rchalo;
		for (j = 0; j < nmax; j++) {
			//set the position
			r = rad[j];
			x = r / rchalo;
			//calculate the local rho based
			rho_tmp = alphahalo / (2 * pow(sqrtpi, 3)) * (exp(pow(-x, 2)) / (pow(x, 2) + pow(qhalo, 2)));
			//renormalize for the new halo size
			rho_tmp = rho_tmp / (rchalo * rchalo * rchalo);
			//calculate mass in local shell, and update total mass
			//dm = rho_tmp * 4 * pi * x * x *dx
			dm = rho_tmp * 4 * pi * r * r *dr;
			mtot = mtot + dm;
			//store values in an array
			rho_halo[j] = rho_tmp * mhalo;
			mass_halo[j] = mtot * mhalo;
		}
		//////////
		//now calculate the potential
		for (j = 0; j < nmax; j++) {
			r = rad[j];
			m = mass_halo[j];
			p1 = -G * m / r;
			phi[j] = p1;
		}
		//////////////////////////////////////////////////////////////////////////////////////////////
		//disk model
		//////////
		//loop over the distribution
		for (j = 0; j < nmax; j++) {
			//set the radius
			rd = rad[j];
			//find the local density in the disk
			rho_local = exp(-rd / hdisk) / (8 * pi*pow(hdisk, 2.0));
			rho_disk[j] = rho_local;
			//find the mass in the spherical shell
			mnew = 4 * pi * rho_local * rd *rd * dr;
			mass_disk[j] = mnew + mold;
			mold = mass_disk[j];
		}
		//////////////////////////////////////////////////////////////////////////////////////////////
		//bulge model
		//////////
		//loop over the distribution
		mold = 0.0;
		for (j = 0; j < nmax; j++) {
			//set the radius
			rd = rad[j];
			//find the local density in the disk
			rho_local = exp(pow(-rd, 2) / pow(hbulge, 2));
			rho_bulge[j] = rho_local;
			//find the mass in the spherical shell
			mnew = 4 * pi * rho_local * rd *rd * dr;
			mass_bulge[j] = mnew + mold;
			mold = mass_bulge[j];
		}
		//renormalize distribution
		factor = mbulge / mass_bulge[nmax];
		for (j = 0; j < nmax; j++) {
			mass_bulge[j] = mass_bulge[j] * factor;
			rho_bulge[j] = rho_bulge[j] * factor;
		}
		dr = rad[2] - rad[1];
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		j = 1;
		mass_total[j] = (mass_halo[j] + mass_disk[j] + mass_bulge[j]);
		r = rad[j];
		rho_total[j] = mass_total[j] / (4.0 / 3.0 * pi * r * r * r);
		dr = rad[2] - rad[1];
		for (j = 1; j < nmax; j++) {
			r = rad[j];
			mass_total[j] = (mass_halo[j] + mass_disk[j] + mass_bulge[j]);
			dm = mass_total[j] - mass_total[j - 1];
			rho_total[j] = dm / (4 * pi * r * r * dr);
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//find the velocity dispersion pow(v_r,2)
		//masses = mass_total;
		//radius = rad;
		//density = rho_total;

		memcpy(masses, mass_total, 4 * nnn);
		memcpy(radius, rad, 4 * nnn);
		memcpy(density, rho_total, 4 * nnn);

		for (j = 0; j < nmax; j++) {
			p = 0.0;
			rr = radius[j];
			dr = radius[nmax] / nmax;
			for (jj = j; jj < nmax; jj++) {
				m = masses[jj];
				rh = density[jj];
				rr = rr + dr;
				dp = rh * G * m / pow(rr, 2) * dr;
				p = p + dp;
			}
			vr2[j] = 1 / density[j] * p;
			vr[j] = sqrt(vr2[j]);
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//find the velocity dispersion pow(v_r,2)
		//masses = mass_total;
		//radius = rad;
		//density = rho_total;

		memcpy(masses, mass_total, 4 * nnn);
		memcpy(radius, rad, 4 * nnn);
		memcpy(density, rho_total, 4 * nnn);

		for (j = 0; j < nmax; j++) {
			p = 0.0;
			rr = radius[j];
			dr = radius[nmax] / nmax;
			for (jj = j; jj < nmax; jj++) {
				m = masses[jj];
				rh = density[jj];
				rr = rr + dr;
				dp = rh * G * m / pow(rr, 2) * dr;
				p = p + dp;
			}
			vr2[j] = 1 / density[j] * p;
			vr[j] = sqrt(vr2[j]);
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//find the accelerations felt by the particles and center of mass
		//masses = mass_total;
		//radius = rad;
		//density = rho_total;

		memcpy(masses, mass_total, 4 * nnn);
		memcpy(radius, rad, 4 * nnn);
		memcpy(density, rho_total, 4 * nnn);

		for (j = 0; j < nmax; j++) {
			rr = radius[j];
			m = masses[j];
			acceleration[j] = G * m / pow(rr, 2);
		}
		//acceleration_particle = acceleration;
		memcpy(acceleration_particle, acceleration, 4 * nnn);

		nmerge = 50;
		acc_merge = acceleration[nmerge];
		rad_merge = rad[nmerge];
		for (j = 0; j < nmerge; j++) {
			rr = radius[j];
			m = masses[j];
			//smoothed acceleration
			acc = G * m / (pow(rr, 2) + .1* (rad_merge - rr));
			acceleration_particle[j] = acc;
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//rederive the masses from the new particle acceleration
		//radius = rad;
		memcpy(radius, rad, 4 * nnn);

		dr = rad[2] - rad[1];
		//find the accelerations felt by the particles and center of mass
		memcpy(radius, rad, 4 * nnn);

		for (j = 0; j < nmax; j++) {
			rr = radius[j];
			new_mass[j] = pow(rr, 2) * acceleration_particle[j] / G;
			new_rho[j] = new_mass[j] / (4 * pi * rr * rr * dr);
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//find the velocity dispersion pow(v_r,2) using the new density and masses


		//masses = new_mass;
		//radius = rad;
		//density = new_rho;

		memcpy(masses, new_mass, 4 * nnn);
		memcpy(radius, rad, 4 * nnn);
		memcpy(density, new_rho, 4 * nnn);

		for (j = 0; j < nmax; j++) {
			p = 0.0;
			rr = radius[j];
			dr = radius[nmax] / nmax;
			for (jj = j; jj < nmax; jj++) {
				m = masses[jj];
				rh = density[jj];
				rr = rr + dr;
				dp = rh * G * m / pow(rr, 2) * dr;
				p = p + dp;
			}
			new_vr2[j] = 1 / density[j] * p;
			new_vr[j] = sqrt(new_vr2[j]);
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//extend the values to large rmax
		for (j = nmax + 1; j < ntotal; j++) {
			mass_total[j] = mass_total[nmax];
			mass_halo[j] = mass_halo[nmax];
			mass_disk[j] = mass_disk[nmax];
			mass_bulge[j] = mass_bulge[nmax];
			new_mass[j] = new_mass[nmax];
			//rho_total[j] = 1e-3;
			//new_rho[j] = new_rho[nmax];
			rho_total[j] = 0.0;
			new_rho[j] = 0.0;
			vr[j] = 1e-6;
			vr2[j] = 1e-6;
			new_vr[j] = 1e-6;
			new_vr2[j] = 1e-6;
			m = mass_total[nmax];
			rr = rad[nmax] + dr*(j - nmax);
			rad[j] = rr;
			acc = G * m / pow(rr, 2);
			acceleration_particle[j] = acc;
			acceleration[j] = acc;
		}
		////////////////////////////////////////////////////////////////////////////////////////////
		//normalize to the unit mass
		for (j = 0; j < ntotal; j++) {
			mass_total[j] = mass_total[j] / 7.13333;
			mass_halo[j] = mass_halo[j] / 7.13333;
			mass_disk[j] = mass_disk[j] / 7.13333;
			mass_bulge[j] = mass_bulge[j] / 7.13333;
			new_mass[j] = new_mass[j] / 7.13333;
			rho_total[j] = rho_total[j] / 7.13333;
			new_rho[j] = new_rho[j] / 7.13333;
			vr[j] = vr[j] / 7.13333;
			vr2[j] = vr2[j] / 7.13333;
			new_vr[j] = new_vr[j] / 7.13333;
			new_vr2[j] = new_vr2[j] / 7.13333;
			rad[j] = rad[j];
			acceleration_particle[j] = acceleration_particle[j] / 7.13333;
			acceleration[j] = acceleration[j] / 7.13333;
			////write(11, *) rad[j], new_rho[j], new_mass[j], new_vr[j];
		}
		pscale = 1.0;
	}
	__device__ int df_index(double rin, double rs)
	{//                       why is rs here?
		double rmax_scale = 100.0;
		int local_nnn = nnn;
		int first_param = int((rin * pscale * rs_internal / rmax_scale) * nnn + 1);
		return min(first_param, local_nnn);
	}
};
class setup_module
{
public:
	df_module* df;
	double* t0;
	double phi_i1, phi_i2, theta_i1, theta_i2;
	vec rscale_i1, rscale_i2;
	double rrout1, rrout2;
	parameters *p;

	__device__ void wrap_rk41(pos_vel xx0, pos_vel& xxe)
	{
		if (p->potential_type == 0)
			xxe = rk41(xx0, &setup_module::diffq_spm);
		else if (p->potential_type == 1)
			xxe = rk41(xx0, &setup_module::diffq_nbi);
		else if (p->potential_type == 2)
			xxe = rk41(xx0, &setup_module::diffq_mond);
	}
	__device__ void perturber_position(pos_vel& original_rv)
	{
		double en, v1;
		pos_vel xx0;
		double omega, incl;
		double epsilon1, epsilon2;

		epsilon1 = sqrt(p->galaxy1.eps);
		epsilon2 = sqrt(p->galaxy2.eps);

		//change inclination and omega into radians
		incl = p->inclination_degree * M_PI / 180.0;
		omega = p->omega_degree * M_PI / 180.0;

		//energy from mass1
		if (p->galaxy1.epsilon > 0.0)
			en = p->galaxy1.mass / epsilon1 * (M_PI / 2.0 - atan(p->rmin / epsilon1));
		else
			en = p->galaxy1.mass / p->rmin;

		//energy from mass2
		if (p->galaxy2.epsilon > 0.0)
			en = p->galaxy2.mass / epsilon2 * (M_PI / 2.0 - atan(p->rmin / epsilon2));
		else
			en = p->galaxy2.mass / p->rmin;

		//calculate escape velocity and velocity at rmin
		v1 = sqrt(2.0 * en);
		v1 = sqrt(2.0)*circular_velocity(p->galaxy1.mass + p->galaxy2.mass, p->rmin,
			rrout1, p->galaxy1.epsilon, p->potential_type, p->a0);

		//adjust velocity for MOND
		v1 = -v1 * p->velocity_factor;


		//setup the transformaton based on the matrix in
		//fundimentals of astrodynamics p-> 82 by
		//bates, mueller, and white(1971)

		xx0.r.x = cos(omega) * p->rmin;
		xx0.r.y = sin(omega) * cos(incl) * p->rmin;
		xx0.r.z = sin(omega) * sin(incl) * p->rmin;

		xx0.v.x = -sin(omega) * v1;
		xx0.v.y = cos(omega) * cos(incl) * v1;
		xx0.v.z = cos(omega) * sin(incl) * v1;

		//update sec_vec
		p->sec_vec = xx0;
		p->sec_vec.v = -p->sec_vec.v;

		perturber_position_vec(xx0, original_rv);
	}
	__device__ void perturber_position_vec(pos_vel xx0, pos_vel& original_rv)
	{
		pos_vel xxe;
		int i;
		double dist1;
		double tcurrent;
		double epsilon1, epsilon2;

		epsilon1 = p->galaxy1.epsilon;
		epsilon2 = p->galaxy2.epsilon;

		//copy the original input vector
		original_rv = xx0;

		//reverse the velocity for backward integration 
		xx0.v = -xx0.v;

		//cout << xx0.print() << endl;

		//now move position back to t0 from t = 0.0
		tcurrent = 0;
		while (*t0 < tcurrent)
		{
			wrap_rk41(xx0, xxe);
			dist1 = xx0.r.dist();
			xx0 = xxe;
			tcurrent = tcurrent - p->h;
		}

		//reverse the velocity for forward integration 
		xx0.v = -xx0.v;

		//now adjust the test particles from the
		//second disk to the proper velocity and positions
		if (p->n > p->galaxy1.n)
			for (i = p->galaxy1.n; i < p->n; i++)
				p->x0[i] = p->x0[i] + xx0;// x0(i, :) = x0(i, :) + xx0(:);

										  //include the perturbing galaxy
										  //p->n += 1;
		p->x0[p->n] = xx0;
	}
	__device__ void reset_perturber_position(pos_vel pv, pos_vel& minloc, pos_vel& zcrossloc, double& tzcross)
	{
		pos_vel xx0, xxe;
		int i, istep;
		double tcurrent, dist, dist_old, zdist, zdist_old;
		bool zmin_flag, min_flag;
		int izmin;
		double rtime;

		//set the positions and velocity of the companion
		xx0.r = pv.r;
		xx0.v = -pv.v;

		//now move position back to t0 from t=0.0
		tcurrent = 0.0;
		p->tend = 0.0;
		istep = 0;
		rtime = 0.0;

		dist_old = 1.0e10;
		zdist_old = 1.0e10;
		min_flag = true;
		zmin_flag = true;

		while (*t0 < tcurrent)
		{

			wrap_rk41(xx0, xxe);

			//write(18, *) xx0

			dist = xx0.r.dist();
			xx0 = xxe;


			//if the distance is larger than the last step, update the
			//clock
			if (dist > dist_old)
			{
				tcurrent = tcurrent - p->h;

				//record the minimum location
				if (min_flag)
					minloc = xx0;
				min_flag = false;

			}
			else
			{
				//if the distance is larger thant he last step, update the
				//ending time of the simulation and the closest point
				p->tend += p->h;
				dist_old = dist;
			}


			//if the distance from the z plan is larger than the last
			//step, set the crossing location and time
			zdist = xx0.r.z;
			if (abs(zdist) > abs(zdist_old) && zmin_flag)
			{
				zmin_flag = false;
				zcrossloc = xx0;
				izmin = istep;
			}
			zdist_old = zdist;

			rtime = rtime + p->h;
			istep = istep + 1;

		}
		tzcross = p->tend - p->h * izmin;

		cout << "t0, tcurrent " << t0 << tcurrent;
		cout << "tend , istep " << p->tend << istep;
		cout << "rtime " << rtime;

		//set the time to t0
		xx0.v = -xx0.v;


		//now move adjust the test particles from the
		//second disk to the proper velocity and positions

		if (p->n > p->galaxy1.n)
			for (i = p->galaxy1.n + 1; i <= p->n; i++)
				p->x0[p->n] = p->x0[p->n] + xx0;

		//include the perturbing galaxy
		p->n += 1;
		p->x0[p->n] = xx0;

		///pscale = 1.1
	}
	__device__ pos_vel rk41(pos_vel xx0, pos_vel(setup_module::*diffq1)(pos_vel))
	{
		pos_vel x, f;
		pos_vel xxe;
		x = xx0;
		f = (this->*diffq1)(x);

		xxe = xx0 + f * p->h / 6.0;
		x = xx0 + f * p->h / 2.0;
		f = (this->*diffq1)(x);

		xxe = xxe + f * p->h / 3.0;
		x = xx0 + f * p->h / 2.0;
		f = (this->*diffq1)(x);

		xxe = xxe + f * p->h / 3.0;
		x = xx0 + f * p->h;
		f = (this->*diffq1)(x);

		xxe = xxe + f * p->h / 6.0;

		return xxe;
	}
	__device__ pos_vel diffq_spm(pos_vel x)
	{
		pos_vel r;
		double r21, r1, a1;
		r21 = x.r.sqrd();
		r1 = sqrt(r21);

		a1 = -p->galaxy1.mass / (r21 + p->galaxy1.eps) - p->galaxy2.mass / (r21 + p->galaxy2.eps);
		r.r = x.v;
		r.v = x.r / r1 * a1;

		return r;
	}
	__device__ pos_vel diffq_nbi(pos_vel x)
	{
		pos_vel r;
		double r21, r1, a1, a2, at;

		double c1, c2, c3, v21, v1, xvalue;
		double sqrtpi;

		int ival, ival2;
		double df_force1, df_force2;
		double df_sigma, df_rho;
		double ee1, ee2;

		//fix to eliminate a compilation warning message for unused variables
		ee1 = p->galaxy1.eps;
		ee2 = p->galaxy2.eps;

		sqrtpi = sqrt(M_PI);

		r21 = x.r.sqrd();
		r1 = sqrt(r21);

		//get the index for the calculations
		ival = df->df_index(r1, rrout1);
		ival2 = df->df_index(r1, rrout2);

		//get the forces, sigma and rho, and rescale them
		df_force1 = df->acceleration_particle[ival] * df->rs2;
		df_force2 = df->acceleration_particle[ival2] * df->rs2;

		df_sigma = df->new_vr[ival] * df->rs2;
		df_rho = df->new_rho[ival] * df->rs3;

		//interpolated forces 
		a1 = -p->galaxy1.mass * df_force1;
		a2 = -p->galaxy2.mass * df_force2;
		at = a1 + a2;

		//df
		v21 = x.v.sqrd();
		v1 = sqrt(v21);
		xvalue = v1 / df_sigma;
		c1 = erf(xvalue) - 2.0 * xvalue / sqrtpi * exp(-xvalue*xvalue);

		//df formula with G=1
		c2 = -4.0 * M_PI * p->galaxy2.mass * df->lnl / v21;
		c3 = c1 * c2 * df_rho;

		r.r = x.v;
		r.v = x.r / r1*at - x.v / v1*c3;

		return r;
	}
	__device__ pos_vel diffq_mond(pos_vel x)
	{
		pos_vel r;
		double r21, r1, a1, tmp, a2;

		r21 = x.r.sqrd();
		r1 = sqrt(r21);

		a1 = -p->galaxy1.mass / (r21 + p->galaxy1.eps);
		a2 = -p->galaxy2.mass / (r21 + p->galaxy2.eps);

		tmp = 2 * p->a0 / a1;
		a1 = a1 / sqrt(2.0) * sqrt(1.0 + sqrt(1.0 + tmp*tmp));

		tmp = 2 * p->a0 / a2;
		a2 = a2 / sqrt(2.0) * sqrt(1.0 + sqrt(1.0 + tmp*tmp));

		a1 = a1 + a2;

		r.r = x.v;
		r.v = x.r / r1*a1;

		return r;
	}
	__device__ void profile()
	{
		profile_g(p->galaxy1, 0);
		profile_g(p->galaxy2, p->galaxy1.n);


	}
	__device__ void profile_g(gparam g, int nstart)
	{
		//variables -
		//opt - option for the distribution
		//rin - inner radius
		//rout - outer radius
		//rscale - scale of brightness drop
		//nstart - start number for placement of particles
		//ntot - number of particles to be placed
		//heat - heat parameter
		//m - mass of galaxy
		//sl - softening length
		//nring - number of rings
		//npart - number of particle per ring(opt)
		//x0 - position of center of mass

		double stheta, ctheta, sphi, cphi;
		double x3, y3, z3, xv3, yv3, zv3, x2, y2, z2, xv2, yv2, zv2;
		double x, y, z, xv, yv, zv;
		int i, j, n;
		double rnorm;
		//double* rp, *r, *angle, *v, *p_ring, *cp_ring, *n_ring;
		double st, ct, dr, ran, r1, r2, ptot;
		int nring, dnring, is, ie, iring, tct;

		double ntot = nstart + g.n;

		n = p->n;
		double r[Gn], angle[Gn], v[Gn];

		stheta = sin(g.theta*M_PI / 180.0);
		ctheta = cos(g.theta*M_PI / 180.0);
		sphi = sin(g.phi*M_PI / 180.0);
		cphi = cos(g.phi*M_PI / 180.0);

		//set up the probablity distribution for the disk

		const int nprof = 1000;
		nring = nprof / 10;

		dnring = nprof / nring;
		double rp[nprof], n_ring[nprof], p_ring[nprof], cp_ring[nprof];

		//set the differential sum of the probability function into a vector
		rnorm = 0.0;
		dr = (g.rout - g.rin) / float(nprof);
		for (i = 0; i < nprof; i++)
		{
			r1 = float(i)*dr + g.rin;
			rp[i] = distrb(r1, g.opt, g.rscale) * r1 * dr * 2.0 * M_PI;
			rnorm = rnorm + rp[i];
		}
		//normalize the vector
		for (i = 0; i < nprof; i++)
			rp[i] /= rnorm;

		//take the fine bins and put them into the selection bins
		tct = 0;
		for (iring = 0; iring < nring; iring++)
		{
			is = (iring - 1) * dnring + 1;
			ie = (iring)* dnring;
			ptot = 0.0;
			for (i = is; i <= ie; i++)
				ptot += rp[i];
			p_ring[iring] = ptot;
		}

		//formulative cumulative distribution function
		cp_ring[0] = p_ring[0];
		for (iring = 1; iring < nring; iring++)
			cp_ring[iring] = cp_ring[iring - 1] + p_ring[iring];

		//find the number of particles in each bin
		memset(n_ring, 0, 4 * nprof);
		//n_ring = 0;

		//cout << "jw nstart = " << nstart << endl;
		//cout << "jw ntot = " << ntot << endl;
		for (i = nstart; i < ntot; i++)
		{
			ran = randm();
			j = 1;
			while (ran > cp_ring[j] && j < nring)
				j = j + 1;
			n_ring[j]++;
		}

		tct = 0;
		i = nstart;
		for (iring = 0; iring <= nring; iring++)
		{
			is = (iring - 1) * dnring + 1;
			ie = (iring)* dnring;
			r1 = float(is)*dr + g.rin;
			r2 = float(ie)*dr + g.rin;
			for (j = 0; j < n_ring[iring]; j++)
			{
				ran = randm();
				r[i] = r1 + ran * (r2 - r1);
				i++;
			}
		}

		//set the angular positions and orbital velocities
		for (i = nstart; i < ntot; i++)
		{
			angle[i] = 2.0 * M_PI * randm();
			v[i] = circular_velocity(g, r[i], p->potential_type, p->a0);
		}

		for (i = nstart; i < ntot; i++)
		{
			st = sin(angle[i]);
			ct = cos(angle[i]);

			x = ct*r[i];
			y = st*r[i];
			z = 0.0;

			xv = -v[i] * st;
			yv = v[i] * ct;
			zv = 0.0;

			x2 = x * ctheta + z * stheta;
			y2 = y;
			z2 = -x * stheta + z * ctheta;
			xv2 = xv * ctheta + zv * stheta;
			yv2 = yv;
			zv2 = -xv * stheta + zv * ctheta;

			x3 = x2  * cphi - y2 * sphi;
			y3 = x2  * sphi + y2 * cphi;
			z3 = z2;
			xv3 = xv2 * cphi - yv2 * sphi;
			yv3 = xv2 * sphi + yv2 * cphi;
			zv3 = zv2;

			p->x0[i].r.x = x3;
			p->x0[i].r.y = y3;
			p->x0[i].r.z = z3;
			p->x0[i].v.x = xv3 + randm()*g.heat;
			p->x0[i].v.y = yv3 + randm()*g.heat;
			p->x0[i].v.z = zv3 + randm()*g.heat;

			//cout << i << "--- " << v[i] << " - " << p->x0[i].print() << endl;
		}
		//delete[] rp, r, angle, v, p_ring, cp_ring, n_ring;
	}
	__device__ double distrb(double r1, double opt, vec rscale)
	{
		if (opt == 1)
			return 1.0 / r1;
		else if (opt == 2)
			return exp(-r1 / rscale.x);
		else if (opt == 3)
			return exp(-r1*r1*rscale.x - rscale.y*r1 - rscale.z);
		return 0;
	}
	__device__ double randm()
	{
		return (double)hiprand_uniform(&(p->state));
	}
	__device__ double circular_velocity(gparam g, double r, int pot, double a0)
	{
		return circular_velocity(g.mass, r, g.rout, g.eps, pot, a0);
	}
	__device__ double circular_velocity(double mass, double r, double rout, double eps, int pot, double a0)
	{
		double ftotal, tmp;
		int ival;
		if (pot == 0)
			ftotal = mass / (r*r + eps);
		else if (pot == 1)
		{
			ival = df->df_index(r, rout);
			ftotal = mass * df->acceleration_particle[ival] * df->rs2;
		}
		else if (pot == 2)
		{
			ftotal = mass / (r*r + eps);
			tmp = 2 * a0 / ftotal;
			ftotal = ftotal / sqrt(2.0) * sqrt(1.0 + sqrt(1.0 + tmp*tmp));
		}
		return sqrt(ftotal * r);
	}
	__device__ void set_perturber_position(pos_vel pv, double t0, pos_vel*x0, int n1, int n)
	{
		pos_vel xx0;
		int i;
		double tcurrent;

		xx0 = pv;
		tcurrent = t0;

		//now move adjust the test particles from the
		//second disk to the proper velocity and positions

		if (n>n1)
			for (i = n1 + 1; i < n; n++)
				x0[i] = x0[i] + xx0;

		//include the perturbing galaxy
		n++;
		x0[n] = xx0;
	}
	__device__ coe rvToCoe(pos_vel pv, double mu)
	{
		vec h, n, v1, v2, ev;
		vec k(0, 0, 1);
		coe r;
		double muInv, rmag, vmag, hmag, nmag, tmp1, tmp2, p, ecc, cosi, cosO, cosw, cosv, cosu;
		muInv = 1.0 / mu;

		rmag = pv.r.dist();
		vmag = pv.v.dist();

		h = pv.r.cross(pv.v);
		hmag = h.dist();

		n = k.cross(h);
		nmag = n.dist();

		tmp1 = vmag*vmag - mu / rmag;
		tmp2 = pv.r.dot(pv.v);

		v1 = pv.r.scale(tmp1);
		v2 = pv.v.scale(tmp2);

		ev = v1 - v2;
		ev = ev.scale(muInv);

		p = hmag*hmag*muInv;
		ecc = ev.dist();
		cosi = h.z / hmag;
		cosO = n.x / nmag;
		cosw = n.dot(ev) / (nmag*ecc);
		cosv = ev.dot(pv.r) / (ecc*rmag);
		cosu = n.dot(pv.r) / (nmag*rmag);

		r.e = p;
		r.a = ecc;
		r.i = acos(cosi);

		tmp1 = acos(cosO);

		if (n.x < 0)
			tmp1 = 2.0*M_PI - tmp1;

		r.o = tmp1;

		tmp1 = acos(cosw);

		if (ev.y < 0)
			tmp1 = 2.0*M_PI - tmp1;

		r.w = tmp1;

		tmp1 = acos(cosv);

		if (pv.r.dot(pv.v) < 0)
			tmp1 = 2.0*M_PI - tmp1;

		r.v = tmp1;
		return r;
	}
	//* Find the time of rmin, assuming earlier than now, given
	//* the r and v values.Returns r and v at time of rmin
	//* by replacing r and v.r and v are given as
	//* {rx, ry, rz, vx, vy, vz}.
	__device__ tmins getTStart(pos_vel rv, double tmin, double mind)
	{
		double t, distOld, distNew, mu, ecc, a, period, apocenter, a2, tApp, distNearApp;
		double minDist, minVel, xxe7, rv7;
		vec r, v;
		pos_vel tmprv, xxe, rvmin;
		tmins outStuff;
		coe coe;
		bool isEllipse;
		mu = p->galaxy1.mass + p->galaxy2.mass;
		t = 0;
		tmprv = rv;
		tmprv.v = -tmprv.v;
		coe = rvToCoe(tmprv, mu);
		ecc = coe.a;
		a = coe.e / (1.0 - ecc*ecc);
		period = 0.0;
		apocenter = a*(1 + ecc);
		a2 = apocenter*apocenter;
		tApp = 0.0;

		isEllipse = false;

		if (ecc < 1.0)
		{
			isEllipse = true;
			period = 2.0 * M_PI / sqrt(mu)*(pow(a, 1.5));
			period = period * 1.0;
		}


		rvmin = rv; //should assign rvmin the values of rv
		rv7 = 0;
		distNew = rv.r.sqrd();
		distOld = 2.0*distNew;

		distNearApp = -1e30;

		//keep looping as long as distance is decreasing
		while (tmin < t)
		{
			coe = rvToCoe(rv, mu);
			xxe7 = t + p->h;
			wrap_rk41(rv, xxe);

			distNew = xxe.r.sqrd();

			//if it's ellipse and it's near apocenter, take this time
			if (isEllipse && (abs(distNew - a2) / a2 < 0.05))
				if (distNew > distNearApp)
				{
					distNearApp = distNew;
					tApp = t;
				}

			if (distNew < distOld)
			{
				distOld = distNew;
				rvmin = xxe;
				rv7 -= p->h;
			}

			rvmin = xxe;
			rv7 = xxe7 - p->h * 2.0;
			t = t - p->h;
		}
		rv = rvmin;

		minDist = rv.r.dist();
		minVel = rv.v.dist();
		t = rv7;

		if (isEllipse && tApp < 0.0)
			t = tApp;
		else
			t = t - mind / minVel;

		outStuff.t = t;
		outStuff.min_dist = minDist;
		outStuff.min_vel = minVel;
		outStuff.rv7 = rv7;

		return outStuff;
	}
};
class integrator {
public:
	pos_vel xe[Gn], f[Gn], x[Gn];
	double r22[Gn], r21[Gn], r2n[Gn];
	double r1[Gn], r2[Gn], rn[Gn];
	double a1[Gn], a2[Gn], a3[Gn];
	double mond_tmp[Gn];

	double m1, m2, m3;
	double eps1, eps2;
	double theta_i1, phi_i1, theta_i2, phi_i2;
	vec rscale_i1, rscale_i2;
	double rrout1, rrout2;

	double df_force11[Gn], df_force22[Gn], df_forcen[Gn], c3n[Gn];
	int ival11[Gn], ival22[Gn], ivaln[Gn];

	int pn, pn1, pn2;

	parameters* p;

	df_module *df;

	__device__ void init_rkvar()
	{
		int n;

		pn = p->n;
		pn1 = p->galaxy1.n;
		pn2 = p->galaxy2.n;

		n = pn + 1;

		//x = new pos_vel[n];
		//f = new pos_vel[n];
		//xe = new pos_vel[n];

		//r22 = new double[n];
		//r21 = new double[n];
		//r2n = new double[n];
		//r1 = new double[n];
		//r2 = new double[n];
		//rn = new double[n];
		//a1 = new double[n];
		//a2 = new double[n];
		//a3 = new double[n];
		//mond_tmp = new double[n];

		m1 = p->galaxy1.mass;
		m2 = p->galaxy2.mass;
		m3 = p->galaxy2.mass;

		eps1 = p->galaxy1.eps;
		eps2 = p->galaxy2.eps;

		//ival11 = new int[n];
		//ival22 = new int[n];
		//ivaln = new int[n];
		//df_force11 = new double[n];
		//df_force22 = new double[n];
		//df_forcen = new double[n];
		//c3n = new double[n];

		phi_i1 = p->galaxy1.phi;
		theta_i1 = p->galaxy1.theta;
		phi_i2 = p->galaxy2.phi;
		theta_i2 = p->galaxy2.theta;

		rscale_i1 = p->galaxy1.rscale;
		rscale_i2 = p->galaxy2.rscale;

		rrout1 = p->galaxy1.rout;
		rrout2 = p->galaxy2.rout;
	}
	void deallocate_rkvar()
	{
		delete x, f, xe;
		delete r22, r21, r2n;
		delete r1, r2, rn;
		delete a1, a2, a3;
		delete mond_tmp;
	}
	__device__ void wrap_rk4()
	{
		if (p->potential_type == 0) {
			rk4(&integrator::diffeq_spm);
		}
		else if (p->potential_type == 1)
			rk4(&integrator::diffeq_nbi);
		else if (p->potential_type == 2)
			rk4(&integrator::diffeq_mond);
	}
	// -------------------------------------------------- -
	// Use this method so that neither the caller of rk4
	// nor implementation need to know which potential
	// is being used
	// -------------------------------------------------- -
	__device__ void rk4(void(integrator::*diffeq)(pos_vel*))
	{
		int n;
		n = p->n;
		memcpy(x, p->x0, sizeof(pos_vel)*n);

		(this->*diffeq)(x);
		//    cout << "lamb!" << endl;
		for (int i = 0; i < n; i++) {
			/*      cout << xe[i].print() << "xe kitty" << endl;
			cout << x[i].print()  <<  "x kitty" << endl;
			cout << f[i].print() << endl;
			*/
			xe[i] = p->x0[i] + f[i] * p->h / 6.0;
			x[i] = p->x0[i] + f[i] * p->h / 2.0;
			/*
			cout << xe[i].print() << "xe kitty poop" << endl;
			cout << x[i].print()  << "x kitty poop" << endl;*/
		}

		(this->*diffeq)(x);
		for (int i = 0; i < n; i++) {
			xe[i] = xe[i] + f[i] * p->h / 3.0;
			x[i] = p->x0[i] + f[i] * p->h / 2.0;
		}

		(this->*diffeq)(x);
		for (int i = 0; i < n; i++) {
			xe[i] = xe[i] + f[i] * p->h / 3.0;
			x[i] = p->x0[i] + f[i] * p->h;
		}

		(this->*diffeq)(x);
		for (int i = 0; i < n; i++) {
			xe[i] = xe[i] + f[i] * p->h / 6.0;
		}

		memcpy(p->xout, xe, sizeof(pos_vel)*n);
		//p->xout = xe;
	}
	__device__ void diffeq_spm(pos_vel *x)
	{
		pos_vel xn;
		int n;

		n = p->n;
		xn = x[n - 1];
		//cerr << x[n].print() << endl;

		for (int i = 0; i < n; i++) {
			r22[i] = pow((x[i].r.x - xn.r.x), 2) + pow((x[i].r.y - xn.r.y), 2) + pow((x[i].r.z - xn.r.z), 2);
			r21[i] = x[i].r.sqrd();
			r2n[i] = xn.r.sqrd();

			r2[i] = sqrt(r22[i]);
			r1[i] = sqrt(r21[i]);
			rn[i] = sqrt(r2n[i]);

			//cout << x[i].print() << endl;
			//cout << x[i].r.x << "duck " << endl;
			//cout << xn.r.y << endl;

			// this is a correction to prevent NaN errors in the vectorized
			// function evalution at the location of the second mass
			r2[n - 1] = 1.0;

			a1[i] = -m1 / (r21[i] + p->galaxy1.eps);
			a2[i] = -m2 / (r22[i] + p->galaxy2.eps);
			a3[i] = -m3 / (r2n[i] + p->galaxy2.eps);

			// calculate the RHS of the diffeq
			f[i].r = x[i].v;

			f[i].v = x[i].r * a1[i] / r1[i] + (x[i].r - xn.r) * a2[i] / r2[i] + xn.r * a3[i] / rn[i];
			//cout << "acc2 = original" << f[i].print() << endl;
			//f[i].v.x = x[i].r.x * a1[i] / r1[i] + (x[i].r.x - xn.r.x) * a2[i] / r2[i] + xn.r.x * a3[i] / rn[i];
			//f[i].v.y = x[i].r.y * a1[i] / r1[i] + (x[i].r.y - xn.r.y) * a2[i] / r2[i] + xn.r.y * a3[i] / rn[i];
			//f[i].v.z = x[i].r.z * a1[i] / r1[i] + (x[i].r.z - xn.r.z) * a2[i] / r2[i] + xn.r.z * a3[i] / rn[i];
			//cout << "acc2 = explicit" << f[i].print() << endl;

			//cout << "acc " <<  a1[i] << ", " << a2[i] << ", " << a3[i] << endl;
			///cout << "acc2 " << f[i].print() << endl;
			//cout << "acc3 " << x[i].r.print() << endl;
			//cout << "acc4 " << xn.r.print() << endl;
			//cout << "acc5 " << r1[i] << ", " << r2[i] << ", " << rn[i] << endl;
			/*      f[i].r.x = 0;
			f[i].r.y = 0;
			f[i].r.z = 0;

			f[i].v.x = 0;
			f[i].v.y = 0;
			f[i].v.z = 0;
			*/

			//cerr << f[i].v.print() << endl;
			//cerr << r1[i] << " " << r2[i] << " " << rn[i] << endl;
		}
	}
	__device__ void diffeq_nbi(pos_vel *x)
	{
		pos_vel xn;
		int n;

		double df_sigma, df_rho;
		double c1, c2, xvalue, v1, v21;
		double sqrtpi;

		sqrtpi = sqrt(M_PI);

		n = p->n;
		xn = x[n];

		for (int i = 0; i < n; i++) {
			// distance between the main galaxy and the particle
			r21[i] = x[i].r.sqrd();
			r1[i] = sqrt(r21[i]);

			// distance between the companion and the particle
			r22[i] = pow((x[i].r.x - xn.r.x), 2) + pow((x[i].r.y - xn.r.y), 2) + pow((x[i].r.z - xn.r.z), 2);
			r2[i] = sqrt(r22[i]);

			// distance between the two galaxies - the tidal force
			r2n[i] = xn.r.sqrd();
			rn[i] = sqrt(r2n[i]);

			ival11[i] = df->df_index(r1[i], rrout1);
			ival22[i] = df->df_index(r2[i], rrout2);
			ivaln[i] = df->df_index(rn[i], rrout2);

			df_force11[i] = df->acceleration_particle[ival11[i]] * df->rs_internal * df->rs_internal;
			df_force22[i] = df->acceleration_particle[ival22[i]] * df->rs_internal * df->rs_internal;
			df_forcen[i] = df->acceleration_particle[ivaln[i]] * df->rs_internal * df->rs_internal;

			// get the forces, sigma and rho, and rescale them
			df_sigma = df->new_vr2[ivaln[1]] * df->rs_internal * df->rs_internal;
			df_rho = df->new_rho[ivaln[1]] * (df->rs_internal * df->rs_internal * df->rs_internal);

			// interpolated forces 
			a1[i] = -m1 * df_force11[i];
			a2[i] = -m2 * df_force22[i];
			a3[i] = -m3 * df_forcen[i];

		}
		// df
		v21 = xn.v.sqrd();
		v1 = sqrt(v21);

		xvalue = v1 / df_sigma;
		c1 = erf(xvalue) - 2.0 * xvalue / sqrtpi * exp(-xvalue*xvalue);

		// df formula with G=1
		c2 = 4.0 * M_PI * m2 * df->lnl / v21;
		memset(c3n, 0, sizeof(double)*n);
		for (int i = pn1; i < n; i++) {
			c3n[i] = c1 * c2 * df_rho;
		}

		// this is a correction to prevent NaN errors in the vectorized
		// function evalution at the location of the second mass
		r2[n] = 1.0;

		// calculate the RHS of the diffeq
		for (int i = 0; i < n; i++) {
			f[i].r = x[i].v;

			f[i].v = x[i].r * a1[i] / r1[i] + (x[i].r - xn.r) * a2[i] / r2[i] + xn.r * a3[i] / rn[i] - xn.v * c3n[i] / v1;
		}
	}
	__device__ void diffeq_mond(pos_vel *x)
	{
		pos_vel xn;
		int n;

		n = p->n;
		xn = x[n];

		for (int i = 0; i < n; i++) {
			r22[i] = pow((x[i].r.x - xn.r.x), 2) + pow((x[i].r.y - xn.r.y), 2) + pow((x[i].r.z - xn.r.z), 2);
			r21[i] = x[i].r.sqrd();
			r2n[i] = xn.r.sqrd();

			r2[i] = sqrt(r22[i]);
			r1[i] = sqrt(r21[i]);
			rn[i] = sqrt(r2n[i]);

			a1[i] = -m1 / (r21[i] + p->galaxy1.epsilon);
			a2[i] = -m2 / (r22[i] + p->galaxy2.epsilon);
			a3[i] = -m3 / (r2n[i] + p->galaxy2.epsilon);

			// this is a correction to prevent NaN errors in the vectorized
			// function evalution at the location of the second mass
			r2[n] = 1.0;


			// scale the accelerations to reflect mond

			mond_tmp[i] = 2 * p->a0 / a1[i];
			a1[i] = a1[i] / sqrt(2.0) * sqrt(1.0 + sqrt(1.0 + mond_tmp[i] * mond_tmp[i]));

			mond_tmp[i] = 2 * p->a0 / a2[i];
			a2[i] = a2[i] / sqrt(2.0) * sqrt(1.0 + sqrt(1.0 + mond_tmp[i] * mond_tmp[i]));

			mond_tmp[i] = 2 * p->a0 / a3[i];
			a3[i] = a3[i] / sqrt(2.0) * sqrt(1.0 + sqrt(1.0 + mond_tmp[i] * mond_tmp[i]));


			// calculate the RHS of the diffeq

			f[i].r = x[i].v;
			f[i].v = x[i].r * a1[i] / r1[i] + (x[i].r - xn.r) * a2[i] / r2[i] + xn.r * a3[i] / rn[i];
		}
	}

};
class init_module
{
public:
	parameters p;
	setup_module s;
	df_module df;
	integrator in;

	bool header_on;
	vec projected[Gn];
	pos_vel original_rv;
	string fname;

	int argc;
	char** argv;

	__device__ void default_parameters()
	{
		p.potential_type = 0;
		p.standard_galaxy_both();
		p.test_collision();

		custom_collision();

		//p.x0 = new pos_vel[p.n + 1];
		//p.xout = new pos_vel[p.n + 1];
		//projected = new vec[p.n + 1];
	}
	void print_run()
	{
		p.print_profile(1);
		p.print_profile(2);
		p.print_collision();
	}
	__device__ void create_collision()
	{
		double tmpT;
		pos_vel r4min;
		tmins t;

		df.init_distribution();
		// create the disks
		s.profile();
		// determine if we need to calculate tStart
		if (!p.tIsSet)
		{
			r4min.r = p.sec_vec.r;
			r4min.v = -p.sec_vec.v;
			t = s.getTStart(r4min, -30.0, 10.0*p.galaxy1.rout);

			tmpT = t.t;
			if (tmpT < 12.0)
				tmpT = -5;

			if (abs(tmpT) < p.h)
				tmpT = -5;

			p.tstart = tmpT;
			p.time = p.tstart;
			p.tIsSet = true;
		}

		//set the perturber galaxy position
		if (!p.use_sec_vec)
			s.perturber_position(original_rv);
		else
			s.perturber_position_vec(p.sec_vec, original_rv);
	}
	void create_images()
	{
		p.iout++;
		fname = "a." + to_string(p.iout);
		FILE *fp;
		fp = fopen(fname.c_str(), "w+");
		p.output_particles(fp, header_on);
		fclose(fp);
	}
	__device__ void take_a_step()
	{
		p.h = p.hbase;
		in.wrap_rk4();

		//cout << p.x0[0].print() << endl;
		//cout << p.xout[0].print() << endl;


		memcpy(p.x0, p.xout, sizeof(pos_vel)*Gn);
		//p.x0 = p.xout;
		p.time += p.h;
	}
	__device__ void custom_collision()
	{
		p.tIsSet = false;
		//string shortbuff;

		//// If command line arguments were passed, set them here
		//int narg = argc;
		//if (narg > 1)
		//{
		//	cout << "custom collision ---------";
		//	shortbuff = argv[1];
		//	if (shortbuff.compare("-f") == 0)
		//	{
		//		//grab the filename
		//		shortbuff = argv[2];
		//		ifstream ifs(shortbuff);
		//		p.read_parameter_file(ifs);
		//		ifs.close();
		//	}
		//	else
		//	{
		//		p.parse_state_info_string(shortbuff);
		//		p.potential_type = 1;
		//		p.h = p.hbase;
		//		p.tstart = -5;
		//		p.tend = 0;
		//		p.time = -5;
		//		if (narg > 2)
		//		{
		//			shortbuff = argv[2];
		//			p.tstart = stod(shortbuff);
		//			p.time = p.tstart;
		//			p.tIsSet = true;
		//		}
		//	}
		//}
		//else
		//{
		p.galaxy1.phi = 5.0;
		p.galaxy1.theta = 5.0;
		p.galaxy1.rscale = 1.0;
		p.galaxy1.rout = 1.0;
		p.galaxy1.mass = 1.0;
		p.galaxy1.epsilon = 0.3;
		p.galaxy1.n = G1n;
		p.galaxy1.heat = 0.0;
		p.galaxy1.opt = 1;

		p.galaxy2.phi = 0.0;
		p.galaxy2.theta = 0.0;
		p.galaxy2.rscale = 0.30;
		p.galaxy2.rout = 0.5;
		p.galaxy2.mass = 0.5;
		p.galaxy2.epsilon = 0.3;
		p.galaxy2.n = G2n;
		p.galaxy2.heat = 0.0;
		p.galaxy2.opt = 1;

		p.inclination_degree = 20.0;
		p.omega_degree = 0.0;
		p.rmin = 0.90;
		p.velocity_factor = 0.90;

		p.h = p.hbase;
		p.time = -5;
		p.tstart = p.time;
		p.tIsSet = true;

		//}

		p.n = p.galaxy1.n + p.galaxy2.n;
		p.galaxy1.eps = p.galaxy1.epsilon*p.galaxy1.epsilon;
		p.galaxy2.eps = p.galaxy2.epsilon*p.galaxy2.epsilon;
	}
	__device__ vec rotation_vector(double theta, double phi)
	{
		vec in;

		double stheta, ctheta, sphi, cphi;

		stheta = sin(theta * M_PI / 180.0);
		ctheta = cos(theta * M_PI / 180.0);
		sphi = sin(phi * M_PI / 180.0);
		cphi = cos(phi * M_PI / 180.0);

		in.x = 0.0;
		in.y = 0.0;
		in.z = 1.0;

		return p.rotate_frame(in, stheta, ctheta, sphi, cphi);
	}
	//  omitted cross_product and rotate_position... unused... along with rotation_vector
};
class basic_run
{
public:
	init_module init;
	double t0, time_interval;
	int nstep_local;

	__device__ void start(int i, int j) {
		// set the disk parameters
		//srand(time(NULL));
		//hiprand_init(clock64(), i, 0, &init.p.state);

		init.s.t0 = &t0;
		init.s.p = &init.p;
		init.s.df = &init.df;
		init.in.df = &init.df;
		init.in.p = &init.p;

		// set the target parameters
		init.default_parameters();

		double infos[] = { 070, 1.0,27,27 - 1.76016,-0.40892,-4.48686,-0.2624,-0.29988,-0.53499,0.6195,0.25405,0.75273,0.45032,88.9,91.3,334.8,0.0,0.3,0.3,0.0,0.0,0.0,0.0,0.0,0.0,1.0,0,0,0.94579,-5.23999,0.76974,0.0,1.0,1.0,0.0,0.0,0.0 };
		init.p.set_state_info(infos);
		init.p.galaxy1.theta = theta_min + theta_step + i;
		init.p.galaxy1.phi = phi_min + phi_step + j;
		//init.p.galaxy1.theta = theta_min + (theta_max - theta_min)*hiprand_uniform(&(init.p.state));
		//init.p.galaxy1.phi = phi_min + (phi_max - phi_min)*hiprand_uniform(&(init.p.state));
		init.create_collision();



		//
		//---- - loop over the system for the output
		//

		//initialize rk routine
		init.in.init_rkvar();

		t0 = init.p.tstart;

		init.p.nstep = (int)((init.p.tend - t0) / init.p.h) + 2;
		nstep_local = init.p.nstep;

		time_interval = (init.p.tend - t0) * 2;

		//init.p.octave_parameters_out(init.original_rv, init.p.x0[init.p.n]);

		//cout << init.original_rv.print() << endl;

		//FILE *fp1 = fopen("initial.txt", "w+");
		//init.p.output_particles(fp1, init.header_on);
		//fclose(fp1);

		//main integration loop

		init.p.iout = 0;

		//nstep_local = 1;

		for (init.p.istep = 0; init.p.istep < nstep_local; init.p.istep++)
		{
			init.take_a_step();
			if (init.p.istep % 50 == 5)
			{
				//printf("Step:\t%d\n", init.p.istep);
				//cerr << init.p.istep << endl;
			}
		}

		//call CREATE_IMAGES
		//init.fname = "a.101";
		//FILE *fp;
		//fp = fopen(init.fname.c_str(), "w+");
		//init.p.output_particles(fp, init.header_on);
		//fclose(fp);

		//this creates a simple script for animating the output with gnuplot
		//if (!init.header_on)
		//	init.p.create_gnuplot_script();

		//delete init.p.x0;
		//delete init.p.xout;
		//init.in.deallocate_rkvar();
	}
};

//hipError_t basic_run_cuda(basic_run *runs, unsigned int size);
__global__ void addKernel(basic_run *runs)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.x;
	int j = blockIdx.x;
	//runs[i].init.argc = 0;
	runs[index].start(i,j);
	printf("Thread %d:%d,%d Done.\n", index, i, j);
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t basic_run_cuda(basic_run *runs, unsigned int size)
{
	basic_run *dev_runs;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	else
		printf("Set Cuda Device\n");

	// Allocate GPU buffers for runs.
	cudaStatus = hipMalloc((void**)&dev_runs, size * sizeof(basic_run));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	else
		printf("Allocate runs array\n");

	// Copy run classes from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_runs, runs, size * sizeof(basic_run), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	else
		printf("Copied runs array\n");

	// Launch a kernel on the GPU with one thread for each element.
	printf("Launching Runs\n");
	addKernel <<< 36, 72 >>>(dev_runs);
	printf("Finished Runs\n");

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else
		printf("No Kernel Errors\n");

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	else
		printf("Cuda Device Syncronized\n");

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(runs, dev_runs, size * sizeof(basic_run), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	else
		printf("Copied runs array back\n");

Error:
	hipFree(dev_runs);

	return cudaStatus;
}

int main()
{
	clock_t begin = clock();

	const int arraySize = 36*72;
	basic_run *runs = new basic_run[arraySize];

	// Add vectors in parallel.
	hipError_t cudaStatus = basic_run_cuda(runs, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "basic_run_cuda failed!");
		system("pause");
		return 1;
	}
	else
		printf("Kernel Run Success\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		system("pause");
		return 1;
	}

	printf("Writing Output Files...");
	for (int i = 0; i < arraySize; i++)
	{
		//call CREATE_IMAGES
		runs[i].init.fname = "a." + to_string(i);
		FILE *fp;
		fp = fopen(runs[i].init.fname.c_str(), "w+");
		runs[i].init.p.output_particles(fp, true);
		fclose(fp);
	}
	printf(" Done\n");

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Time: %f\n", time_spent);
	system("pause");
	return 0;
}
